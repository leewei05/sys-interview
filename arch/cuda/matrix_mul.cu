#include <hip/hip_runtime.h>

__global__ void matrix_multiplication_kernel(const float* A, const float* B, float* C, int M, int N, int K) {
    int col = blockIdx.x * blockDim.x + threadIdx.x; // col
    int row = blockIdx.y * blockDim.y + threadIdx.y; // row

    // Boundary check because the grid was sized using ceiling division
    // A = M * N
    // B = N * K
    // Think of the final matrix M by K, which means there are M rows and K columns
    if (col < K && row < M) {
        float tmp = 0;
        // Each term in the final matrix is calculated N times
        for(int i =0; i< N; i++) {
            // A: iterate row
            // B: iterate col
            tmp += A[row * N + i] * B[i * K + col];
        }
        // C = M * K
        C[row * K + col] = tmp;
    }
}

// A, B, C are device pointers (i.e. pointers to memory on the GPU)
extern "C" void solve(const float* A, const float* B, float* C, int M, int N, int K) {
    dim3 threadsPerBlock(16, 16);
    dim3 blocksPerGrid((K + threadsPerBlock.x - 1) / threadsPerBlock.x,
                       (M + threadsPerBlock.y - 1) / threadsPerBlock.y);
    
    matrix_multiplication_kernel<<<blocksPerGrid, threadsPerBlock>>>(A, B, C, M, N, K);
    hipDeviceSynchronize();
}
